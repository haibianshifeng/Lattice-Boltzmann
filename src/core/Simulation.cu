#include "hip/hip_runtime.h"
#include "core/Simulation.h"

namespace boltzmann {
    namespace core {
        Simulation::Simulation(int width_, int height_) : xdim(width_), ydim(height_) {
            this->xdim = width_;
            this->ydim = height_;

            hipMallocManaged(&this->n0, sizeof(double *) * ydim);
            hipMallocManaged(&this->nN, sizeof(double *) * ydim);
            hipMallocManaged(&this->nS, sizeof(double *) * ydim);
            hipMallocManaged(&this->nE, sizeof(double *) * ydim);
            hipMallocManaged(&this->nW, sizeof(double *) * ydim);
            hipMallocManaged(&this->nNW, sizeof(double *) * ydim);
            hipMallocManaged(&this->nNE, sizeof(double *) * ydim);
            hipMallocManaged(&this->nSW, sizeof(double *) * ydim);
            hipMallocManaged(&this->nSE, sizeof(double *) * ydim);
            hipMallocManaged(&this->density, sizeof(double *) * ydim);
            hipMallocManaged(&this->xvel, sizeof(double *) * ydim);
            hipMallocManaged(&this->yvel, sizeof(double *) * ydim);
            hipMallocManaged(&this->speed2, sizeof(double *) * ydim);
            hipMallocManaged(&this->curl, sizeof(double *) * ydim);
            hipMallocManaged(&this->barrier, sizeof(bool *) * ydim);

            hipMallocManaged(&this->n0_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nN_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nS_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nE_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nW_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nNW_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nNE_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nSW_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->nSE_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->density_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->xvel_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->yvel_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->speed2_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->curl_temp, sizeof(double *) * ydim);
            hipMallocManaged(&this->barrier_temp, sizeof(bool *) * ydim);

            for (int i = 0; i < this->ydim; i++) {
                hipMallocManaged(&this->n0[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nN[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nS[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nE[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nW[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nNW[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nNE[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nSW[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nSE[i], sizeof(double) * xdim);
                hipMallocManaged(&this->density[i], sizeof(double) * xdim);
                hipMallocManaged(&this->xvel[i], sizeof(double) * xdim);
                hipMallocManaged(&this->yvel[i], sizeof(double) * xdim);
                hipMallocManaged(&this->speed2[i], sizeof(double) * xdim);
                hipMallocManaged(&this->curl[i], sizeof(double) * xdim);
                hipMallocManaged(&this->barrier[i], sizeof(barrier) * xdim);

                hipMallocManaged(&this->n0_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nN_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nS_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nE_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nW_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nNW_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nNE_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nSW_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->nSE_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->density_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->xvel_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->yvel_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->speed2_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->curl_temp[i], sizeof(double) * xdim);
                hipMallocManaged(&this->barrier_temp[i], sizeof(barrier) * xdim);

                hipMemset(this->n0[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nN[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nS[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nW[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nE[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nNW[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nNE[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nSW[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nSE[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->density[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->xvel[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->yvel[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->speed2[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->curl[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->barrier[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(bool));

                hipMemset(this->n0_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nN_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nS_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nW_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nE_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nNW_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nNE_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nSW_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->nSE_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->density_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->xvel_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->yvel_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->speed2_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->curl_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(double));
                hipMemset(this->barrier_temp[i], 0, static_cast<unsigned long>(this->xdim) * sizeof(bool));
            }

            hipDeviceSynchronize();

            this->draw_circle(100, 100, 40);
            this->draw_circle(100, 300, 40);
            this->draw_circle(100, 500, 40);
            this->draw_circle(100, 700, 40);
            this->draw_circle(100, 900, 40);

            this->draw_circle(250, 200, 40);
            this->draw_circle(250, 400, 40);
            this->draw_circle(250, 600, 40);
            this->draw_circle(250, 800, 40);

            this->draw_circle(400, 100, 40);
            this->draw_circle(400, 300, 40);
            this->draw_circle(400, 500, 40);
            this->draw_circle(400, 700, 40);
            this->draw_circle(400, 900, 40);

            this->synchronize();
            hipDeviceSynchronize();

            this->init_fluid();
            hipDeviceSynchronize();

            this->synchronize();
            hipDeviceSynchronize();
        }

        Simulation::~Simulation() {
            for (int i = 0; i < this->ydim; i++) {
                hipFree(this->n0[i]);
                hipFree(this->nN[i]);
                hipFree(this->nS[i]);
                hipFree(this->nE[i]);
                hipFree(this->nW[i]);
                hipFree(this->nNW[i]);
                hipFree(this->nNE[i]);
                hipFree(this->nSW[i]);
                hipFree(this->nSE[i]);
                hipFree(this->density[i]);
                hipFree(this->xvel[i]);
                hipFree(this->yvel[i]);
                hipFree(this->speed2[i]);
                hipFree(this->curl[i]);
                hipFree(this->barrier[i]);

                hipFree(this->n0_temp[i]);
                hipFree(this->nN_temp[i]);
                hipFree(this->nS_temp[i]);
                hipFree(this->nE_temp[i]);
                hipFree(this->nW_temp[i]);
                hipFree(this->nNW_temp[i]);
                hipFree(this->nNE_temp[i]);
                hipFree(this->nSW_temp[i]);
                hipFree(this->nSE_temp[i]);
                hipFree(this->density_temp[i]);
                hipFree(this->xvel_temp[i]);
                hipFree(this->yvel_temp[i]);
                hipFree(this->speed2_temp[i]);
                hipFree(this->curl_temp[i]);
                hipFree(this->barrier_temp[i]);
            }
            hipFree(this->n0);
            hipFree(this->nN);
            hipFree(this->nS);
            hipFree(this->nE);
            hipFree(this->nW);
            hipFree(this->nNW);
            hipFree(this->nNE);
            hipFree(this->nSW);
            hipFree(this->nSE);
            hipFree(this->density);
            hipFree(this->xvel);
            hipFree(this->yvel);
            hipFree(this->speed2);
            hipFree(this->curl);
            hipFree(this->barrier);

            hipFree(this->n0_temp);
            hipFree(this->nN_temp);
            hipFree(this->nS_temp);
            hipFree(this->nE_temp);
            hipFree(this->nW_temp);
            hipFree(this->nNW_temp);
            hipFree(this->nNE_temp);
            hipFree(this->nSW_temp);
            hipFree(this->nSE_temp);
            hipFree(this->density_temp);
            hipFree(this->xvel_temp);
            hipFree(this->yvel_temp);
            hipFree(this->speed2_temp);
            hipFree(this->curl_temp);
            hipFree(this->barrier_temp);
        }

        void Simulation::init_fluid() const {
            for (int y = 0; y < ydim; y++) {
                for (int x = 0; x < xdim; x++) {
                    if (barrier[y][x]) {
                        zeroSite(x, y);
                    } else {
                        n0[y][x] = four9ths * (1 - 1.5 * v * v);
                        nE[y][x] = one9th * (1 + 3 * v + 3 * v * v);
                        nW[y][x] = one9th * (1 - 3 * v + 3 * v * v);
                        nN[y][x] = one9th * (1 - 1.5 * v * v);
                        nS[y][x] = one9th * (1 - 1.5 * v * v);
                        nNE[y][x] = one36th * (1 + 3 * v + 3 * v * v);
                        nSE[y][x] = one36th * (1 + 3 * v + 3 * v * v);
                        nNW[y][x] = one36th * (1 - 3 * v + 3 * v * v);
                        nSW[y][x] = one36th * (1 - 3 * v + 3 * v * v);
                        density[y][x] = 1;
                        xvel[y][x] = v;
                        yvel[y][x] = 0;
                        speed2[y][x] = v * v;
                    }
                }
            }
        }

        void Simulation::zeroSite(int x, int y) const {
            n0[y][x] = 0;
            nE[y][x] = 0;
            nW[y][x] = 0;
            nN[y][x] = 0;
            nS[y][x] = 0;
            nNE[y][x] = 0;
            nNW[y][x] = 0;
            nSE[y][x] = 0;
            nSW[y][x] = 0;
            xvel[y][x] = 0;
            yvel[y][x] = 0;
            speed2[y][x] = 0;


            n0_temp[y][x] = 0;
            nE_temp[y][x] = 0;
            nW_temp[y][x] = 0;
            nN_temp[y][x] = 0;
            nS_temp[y][x] = 0;
            nNE_temp[y][x] = 0;
            nNW_temp[y][x] = 0;
            nSE_temp[y][x] = 0;
            nSW_temp[y][x] = 0;
            xvel_temp[y][x] = 0;
            yvel_temp[y][x] = 0;
            speed2_temp[y][x] = 0;
        }

        void Simulation::collide() const {
            boltzmann::core::collide<<<this->xdim,this->ydim>>>(
                    xdim,
                            ydim,
                            barrier,
                            n0,
                            nN,
                            nS,
                            nE,
                            nW,
                            nNW,
                            nNE,
                            nSW,
                            nSE,
                            density,
                            xvel,
                            yvel,
                            speed2,
                            n0_temp,
                            nN_temp,
                            nS_temp,
                            nE_temp,
                            nW_temp,
                            nNW_temp,
                            nNE_temp,
                            nSW_temp,
                            nSE_temp,
                            density_temp,
                            xvel_temp,
                            yvel_temp,
                            speed2_temp,
                            omega);
        }

        void Simulation::stream() const {
            boltzmann::core::stream<<<this->xdim, this->ydim>>>(
                    xdim,
                            ydim,
                            barrier,
                            n0,
                            nN,
                            nS,
                            nE,
                            nW,
                            nNW,
                            nNE,
                            nSW,
                            nSE,
                            density,
                            xvel,
                            yvel,
                            speed2,
                            n0_temp,
                            nN_temp,
                            nS_temp,
                            nE_temp,
                            nW_temp,
                            nNW_temp,
                            nNE_temp,
                            nSW_temp,
                            nSE_temp,
                            density_temp,
                            xvel_temp,
                            yvel_temp,
                            speed2_temp,
                            omega,
                            v);
        }

        void Simulation::bounce() const {
            boltzmann::core::bounce<<<this->xdim,this->ydim>>>(
                            xdim,
                            ydim,
                            barrier,
                            n0,
                            nN,
                            nS,
                            nE,
                            nW,
                            nNW,
                            nNE,
                            nSW,
                            nSE,
                            density,
                            xvel,
                            yvel,
                            speed2,
                            n0_temp,
                            nN_temp,
                            nS_temp,
                            nE_temp,
                            nW_temp,
                            nNW_temp,
                            nNE_temp,
                            nSW_temp,
                            nSE_temp,
                            density_temp,
                            xvel_temp,
                            yvel_temp,
                            speed2_temp,
                            omega,
                            v);
        }

        void Simulation::compute_curl() const {
            boltzmann::core::compute_curl<<<this->xdim, this->ydim>>>(xdim, ydim, curl, yvel, xvel);
        }

        void Simulation::draw_barrier(int x, int y) const {
            barrier[y][x] = true;
            zeroSite(x, y);
        }

        void Simulation::draw_circle(int x_center, int y_center, int radius) const {
            for (int y = 0; y < ydim; y++) {
                for (int x = 0; x < xdim; x++) {
                    if (std::sqrt((x - x_center) * (x - x_center) + (y - y_center) * (y - y_center)) < radius) {
                        draw_barrier(x, y);
                    }
                }
            }
        }

        void Simulation::synchronize() const {
            boltzmann::core::synchronize<<<this->xdim, this->ydim>>>(
                    xdim,
                            ydim,
                            barrier,
                            n0,
                            nN,
                            nS,
                            nE,
                            nW,
                            nNW,
                            nNE,
                            nSW,
                            nSE,
                            density,
                            xvel,
                            yvel,
                            speed2,
                            n0_temp,
                            nN_temp,
                            nS_temp,
                            nE_temp,
                            nW_temp,
                            nNW_temp,
                            nNE_temp,
                            nSW_temp,
                            nSE_temp,
                            density_temp,
                            xvel_temp,
                            yvel_temp,
                            speed2_temp,
                            omega,
                            v);
        }

        void Simulation::debug_information() {
            for (int y = 0; y < ydim; y++) {
                printf("%f ", this->speed2[y][500]);
            }
            printf("\n");
        }
    }
}
