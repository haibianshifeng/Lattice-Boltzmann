#include "hip/hip_runtime.h"
#include "app/GUI.h"

namespace boltzmann {
    namespace app {
        GUI::GUI(sf::Window *render_window_, boltzmann::core::Simulation *simulation_, bool freaky_colors)
                : render_window(render_window_), simulation(simulation_), colorful(freaky_colors) {
            // Allocate memory for OpenGL coordinates
            hipMallocManaged(&coordinates, sizeof(float *) * simulation->ydim);

            // Allocate memory for OpenGL colors
            hipMallocManaged(&pixels, sizeof(uint8_t *) * simulation->ydim);

            /*
             * Initialize memory for OpenGL coordinates and colors
             */
            for (int y = 0; y < this->simulation->ydim; y++) {
                hipMallocManaged(&coordinates[y], sizeof(float) * 2 * simulation->xdim);
                hipMallocManaged(&pixels[y], sizeof(uint8_t) * 3 * simulation->xdim);

                for (int x = 0; x < this->simulation->xdim; x++) {
                    coordinates[y][2 * x] = (float) x;
                    coordinates[y][2 * x + 1] = (float) y;
                    if (this->simulation->barrier[y][x]) {
                        pixels[y][x * 3] = 50;
                        pixels[y][x * 3 + 1] = 50;
                        pixels[y][x * 3 + 2] = 50;
                    }
                }
            }

            /*
             * Initialize rainbow color for the spectrum
             */
            this->setNColors(this->getNColors());
        }

        GUI::~GUI() {
            for (int y = 0; y < this->simulation->ydim; y++) {
                hipFree(&coordinates[y]);
                hipFree(&pixels[y]);
            }
            hipFree(pixels);
            hipFree(colors);
            hipFree(coordinates);
        }

        void GUI::paint(uint32_t mode) {
            /*
             * Rendering the pixels
             * 0: Curl
             * 1: speed
             * 2: x velocity
             * 3: y velocity
             * 4: density
             */
            switch (mode) {
                case 0:
                    boltzmann::core::update_pixels_curl<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->curl,
                            contrast,
                            colors);
                    break;
                case 1:
                    boltzmann::core::update_pixels_speed<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->speed2,
                            contrast,
                            colors);
                    break;
                case 2:
                    boltzmann::core::update_pixels_xvel<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->xvel,
                            contrast,
                            colors);
                    break;
                case 3:
                    boltzmann::core::update_pixels_yvel<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->yvel,
                            contrast,
                            colors);
                    break;
                case 4:
                    boltzmann::core::update_pixels_density<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->density,
                            contrast,
                            colors);
                    break;
                default:
                    boltzmann::core::update_pixels_curl<<<simulation->xdim, simulation->ydim>>>(
                    simulation->ydim,
                            simulation->xdim,
                            pixels,
                            simulation->barrier,
                            n_colors,
                            simulation->curl,
                            contrast,
                            colors);
            }
            hipDeviceSynchronize();

            /*
             * Plotting
             */
            glClearColor(0, 0, 0, 0);
            glClear(GL_COLOR_BUFFER_BIT);

            glPushMatrix();
            glEnableClientState(GL_VERTEX_ARRAY);
            glEnableClientState(GL_COLOR_ARRAY);

            for (uint32_t y = 0; y < this->simulation->ydim; y++) {
                glVertexPointer(2, GL_FLOAT, 0, coordinates[y]);
                glColorPointer(3, GL_UNSIGNED_BYTE, 0, pixels[y]);
                glDrawArrays(GL_POINTS, 0, simulation->xdim);
            }

            glDisableClientState(GL_VERTEX_ARRAY);
            glDisableClientState(GL_COLOR_ARRAY);

            glPopMatrix();
            glFlush();

            render_window->display();
        }

        double GUI::getContrast() const {
            return contrast;
        }

        void GUI::setContrast(double contrast_) {
            GUI::contrast = contrast_;
        }

        int GUI::getNColors() const {
            return n_colors;
        }

        void GUI::setNColors(int nColors) {
            this->n_colors = std::max(1000, nColors);
            this->n_colors = std::min(1000000, this->n_colors);

            if (this->colors) {
                hipFree(colors);
            }

            // Allocate memory for rainbow colors spectrum
            hipMallocManaged(&colors, sizeof(sf::Color) * n_colors);

            if (this->colorful) {
                for (int c = 0; c < n_colors; c++) {
                    double h = (double) c / n_colors;
                    h += 10 * sin(2 * M_PI * h);
                    colors[c] = boltzmann::utils::HSBtoRGB((float) h, 0.75, 1);
                }
            } else {
                for (int c = 0; c < n_colors; c++) {
                    double h = (2.0 / 3) * (1 - c * 1.0 / n_colors);
                    h += 0.03 * sin(6 * M_PI * h);
                    colors[c] = boltzmann::utils::HSBtoRGB((float) h, (float) 1, (float)1);
                }
            }
        }

        bool GUI::isColorful() const {
            return colorful;
        }

        void GUI::setColorful(bool colorful_) {
            GUI::colorful = colorful_;
            this->setNColors(this->getNColors());
        }
    }
}
