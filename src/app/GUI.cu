#include "hip/hip_runtime.h"
#include "core/Simulation.h"
#include "app/GUI.h"

namespace boltzmann {
    namespace app {

        __global__
        void update_pixels(uint32_t ydim, uint32_t xdim, sf::Vertex * pixels, bool ** barrier, uint32_t n_colors, double **curl, double contrast, sf::Color * colors) {
            uint32_t y = blockIdx.x;
            uint32_t x = threadIdx.x;

            if(y < ydim && x < xdim) {
                if (barrier[y][x]) {
                    pixels[y * xdim + x].color.r = 255;
                    pixels[y * xdim + x].color.g = 255;
                    pixels[y * xdim + x].color.b = 255;
                } else {
                    auto colorIndex = min(n_colors - 1,
                                          (int) (n_colors *
                                                 (0.5 + curl[y][x] * contrast * 0.3)));
                    colorIndex = max(0, colorIndex);
                    colorIndex = min(n_colors - 1, colorIndex);
                    pixels[y * xdim + x].color = colors[colorIndex];
                }
            }
        }

        GUI::GUI(sf::RenderWindow *render_window_, boltzmann::core::Simulation *simulation_)
                : render_window(render_window_), simulation(simulation_) {
            pixels = new sf::Vertex[this->simulation->ydim * this->simulation->xdim];
            vertex_buffer.create(static_cast<size_t>(this->simulation->ydim * this->simulation->xdim));
            hipMallocManaged(&colors, sizeof(sf::Color) * n_colors);
            hipMallocManaged(&pixels, sizeof(sf::Vertex) * this->simulation->ydim * this->simulation->xdim);

            int i = 0;
            for (int y = 0; y < this->simulation->ydim; y++) {
                for (int x = 0; x < this->simulation->xdim; x++) {
                    pixels[i++].position = sf::Vector2f{static_cast<float>(x), static_cast<float>(y)};
                }
            }

            sf::Color start_color{0, 0, 0};
            sf::Color end_color{255, 255, 255};
            for (int c = 0; c < n_colors; c++) {
                double percent = (double) c / (double) n_colors;

                auto r = static_cast<uint8_t>((double) start_color.r +
                                              percent * ((double) end_color.r - (double) start_color.r));

                auto g = static_cast<uint8_t>((double) start_color.g +
                                              percent * ((double) end_color.g - (double) start_color.g));

                auto b = static_cast<uint8_t>((double) start_color.b +
                                              percent * ((double) end_color.b - (double) start_color.b));

                colors[c] = sf::Color{r, g, b};
            }
        }

        GUI::~GUI() {
            hipFree(pixels);
            hipFree(colors);
        }

        void GUI::paint() {
            update_pixels<<<simulation->ydim, simulation->xdim>>>(simulation->ydim, simulation->xdim, pixels, simulation->barrier, n_colors, simulation->curl, contrast, colors);
            hipDeviceSynchronize();
            vertex_buffer.update(pixels);
            render_window->draw(vertex_buffer);
            render_window->display();
        }
    }
}