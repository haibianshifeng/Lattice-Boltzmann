#include "hip/hip_runtime.h"
#include "core/Simulation.h"
#include "app/GUI.h"

namespace boltzmann {
    namespace app {
        __global__
        void
        update_pixels(uint32_t ydim, uint32_t xdim, sf::Vertex *pixels, bool **barrier, double n_colors, double **curl,
                      double contrast, sf::Color *colors) {
            uint32_t y = blockIdx.x;
            uint32_t x = threadIdx.x;

            if (y < ydim && x < xdim) {
                if (barrier[y][x]) {
                    pixels[y * xdim + x].color.r = 125;
                    pixels[y * xdim + x].color.g = 125;
                    pixels[y * xdim + x].color.b = 125;
                } else {
                    auto colorIndex = min(n_colors - 1,
                                          (n_colors *
                                           (0.5f + curl[y][x] * contrast * 0.3f)));
                    colorIndex = max(0.0f, colorIndex);
                    colorIndex = min(n_colors - 1, colorIndex);
                    pixels[y * xdim + x].color = colors[(uint32_t) colorIndex];
                }
            }
        }


        GUI::GUI(sf::RenderWindow *render_window_, boltzmann::core::Simulation *simulation_)
                : render_window(render_window_), simulation(simulation_) {
            pixels = new sf::Vertex[this->simulation->ydim * this->simulation->xdim];
            vertex_buffer.create((uint64_t) (this->simulation->ydim * this->simulation->xdim));
            hipMallocManaged(&colors, sizeof(sf::Color) * n_colors);
            hipMallocManaged(&pixels, sizeof(sf::Vertex) * this->simulation->ydim * this->simulation->xdim);

            for (int y = 0; y < this->simulation->ydim; y++) {
                for (int x = 0; x < this->simulation->xdim; x++) {
                    pixels[y * this->simulation->xdim + x].position = sf::Vector2f{static_cast<float>(x),
                                                                                   static_cast<float>(y)};
                }
            }

            sf::Color start_color{0, 0, 0};
            sf::Color end_color{255, 255, 255};
            for (int c = 0; c < n_colors; c++) {
                double percent = (double) c / (double) n_colors;

                auto r = (uint8_t) ((double) start_color.r + percent * ((double) end_color.r - (double) start_color.r));

                auto g = (uint8_t) ((double) start_color.g + percent * ((double) end_color.g - (double) start_color.g));

                auto b = (uint8_t) ((double) start_color.b + percent * ((double) end_color.b - (double) start_color.b));

                colors[c] = sf::Color{r, g, b};
            }
        }

        GUI::~GUI() {
            hipFree(pixels);
            hipFree(colors);
        }

        void GUI::paint() {
            update_pixels<<<simulation->ydim, simulation->xdim>>>(simulation->ydim, simulation->xdim, pixels,
                                                                  simulation->barrier, n_colors, simulation->curl,
                                                                  contrast, colors);
            hipDeviceSynchronize();
            vertex_buffer.update(pixels);
            render_window->draw(vertex_buffer);
            render_window->display();

        }
    }
}